
#include <hip/hip_runtime.h>

__global__ void FWT_SHFL(const float *fi, float *Fa, const int *seq,
        const int Pa, const int Na, const int N)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;    // thread Id

    float F1; // storing last value
    float F2; // will be shuffled, all threads have one

    int seqi; // where in memory to put value

    // calculate whether mem pull will be made neg
    // [0:1] -> [0:2] -> [-1:1] -> [1:-1]
    int negMask;

    // whether to accept shfl this round
    int srcMask;

    // every thread load first piece of data
    if (tid < N) {
        F1 = fi[tid];
    }

    // memory pull hidden by next ops
    seqi = seq[tid];

    int Nm = Na/2;
    for(int pm=0;pm<Pa;pm++) {
        // calculate src mask
        srcMask = ((tid >> (Pa-pm-1)) & 1LU) ^ 1LU; // 0 or 1

        // calculate negMask
        negMask = (((tid >> (Pa-pm-1)) & 1LU) * 2 - 1) * -1;    // 1 or -1

        // apply warp shuffle down
        F2 = srcMask * __shfl_down_sync(0xFFFFFFFF, F1, Nm);

        // flip mask
        srcMask ^= 1LU;

        // apply warp shuffle up
        F2 += srcMask * __shfl_up_sync(0xFFFFFFFF, F1, Nm);

        // add to existing warp value, using negMask
        F1 = F1 * negMask + F2;

        // update shfl width
        Nm >>= 1;
    }

    // write to global memory
    if (tid < N) {
        Fa[(tid / 32) * 32 + seqi] = F1;
    }

    return;
}

